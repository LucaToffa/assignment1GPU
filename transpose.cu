#include <stdio.h>
#include <hip/hip_runtime.h>

#define DEFAULT_SIZE 32
//#define DEBUG
#ifdef DEBUG
    #define PRINTF(...) printf(__VA_ARGS__)
#else
    #define PRINTF(...)
#endif
//#define PRINT
#ifndef TILE_SIZE
    #define TILE_SIZE 32 
#endif
#ifndef BLOCK_ROWS
    #define BLOCK_ROWS 8 //works up to 16
#endif
#define TRANSPOSITIONS 100

//i think there is a problem with the matrix allocation
//try managed memory

int initMatrix(float* mat, int size);
int printMatrix(float* mat, int size);
int testTranspose(float* mat, float* mt, int size);
int block_benchmark(uint N);
int basic_benchmark(uint N);
int conflict_benchmark(uint N);

//implementation of block transpose in gpu
//each kernel is a block?
//decoupling tile size in shares memory and block size not possible
__global__ void block_transpose(float *input, float *output){
    __shared__ float tile[TILE_SIZE][TILE_SIZE+1];
    
    //input to shared offsets
    int x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int y = blockIdx.y * TILE_SIZE + threadIdx.y;
    int w = gridDim.x * TILE_SIZE;

    for(int i = 0; i < TILE_SIZE; i += BLOCK_ROWS){
        tile[threadIdx.y+i][threadIdx.x] = input[(y+i) * w + x];
    }

    __syncthreads();

    //shared to output offsets
    x = blockIdx.y * TILE_SIZE + threadIdx.x;
    y = blockIdx.x * TILE_SIZE + threadIdx.y;

    for(int j = 0; j < TILE_SIZE; j += BLOCK_ROWS){
        output[(y+j) * w + x] = tile[threadIdx.x][threadIdx.y+j];
    }


}

//without the +1 the memory access conflicts cannot be avoided
__global__ void conflict_transpose(float *input, float *output){
    __shared__ float tile[TILE_SIZE][TILE_SIZE];
    
    //input to shared offsets
    int x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int y = blockIdx.y * TILE_SIZE + threadIdx.y;
    int w = gridDim.x * TILE_SIZE;

    for(int i = 0; i < TILE_SIZE; i += BLOCK_ROWS){
        tile[threadIdx.y+i][threadIdx.x] = input[(y+i) * w + x];
    }

    __syncthreads();

    //shared to output offsets
    x = blockIdx.y * TILE_SIZE + threadIdx.x;
    y = blockIdx.x * TILE_SIZE + threadIdx.y;

    for(int j = 0; j < TILE_SIZE; j += BLOCK_ROWS){
        output[(y+j) * w + x] = tile[threadIdx.x][threadIdx.y+j];
    }

}

#define B_TILE TILE_SIZE
#define B_ROWS BLOCK_ROWS
// implementation of basic transpose in gpu
// to avoid ifs in the kernel, check the matrix size and derive block/threads size
__global__ void basic_transpose(float *input, float *output, int N){
    //matrix transpose that works for any size
    int x = blockIdx.x * B_TILE + threadIdx.x;
    int y = blockIdx.y * B_TILE + threadIdx.y;

    int index_in = x + N * y;
    int index_out = y + N * x;

    for (int i = 0; i < B_TILE; i += B_ROWS){
        output[index_out + i] = input[index_in + i * N];
    }
}

int main(int argc, char* argv[]){
#ifdef DEBUG
    if (argc > 1){
        printf("argc = %d:\n", argc);
        for(int i = 0; i < argc; i++){
            printf("arg %d : %s\n", i+1, argv[i]);
        }
        printf("\n");
    }
    
#endif
    bool swipe = false;
    uint N = DEFAULT_SIZE;
    if(argc >= 2){
        N = (1<<atoi(argv[1]));
        PRINTF("N changed: %d\n", N);
        printf("N: %d, T: %d, B: %d \n", N, TILE_SIZE, BLOCK_ROWS);
    }else{
        swipe = true;
        //log shape of data
        printf("#T: %d, B: %d#\n", TILE_SIZE, BLOCK_ROWS);
        printf("#N, OpTime, Op-GB/s, KTime, K-GB/s (basic, conflcit, block)#\n");
    }
    if(BLOCK_ROWS > TILE_SIZE){
        printf("Error: BLOCK_ROWS > TILE_SIZE\n");
        return -1;

    }

    // int mem_size = N * N * sizeof(float);
    // PRINTF("Memory size: %d\n", mem_size);
    // //init the matrix to transpose in gpu
    // float* mat = (float*) malloc(mem_size);

    // float* mat_t = (float*) malloc(mem_size);//for some reason the gpu segfaults if i dont maccoc in the function
    // memset(mat_t, 0, mem_size); 

    // initMatrix(mat, N);
    // printMatrix(mat, N);

    /*
    everything gpu related runs in these functions, a bit repetitive 
    but i dont wont to pass around that many parameters
    */
    do{
        PRINTF("N: ");
        printf("%d, ", N);
        basic_benchmark(N);
        conflict_benchmark(N);
        block_benchmark(N);
        N *= 2;
    }while(swipe && N < (2<<13)); //2<<14 = 16384

    //dealloc local memory memory
    // free(mat);
    // free(mat_t);
    PRINTF("\n");
    return 0;
}


int initMatrix(float* mat, int size){
    for(int i = 0; i < size; i++){
        for(int j = 0; j < size; j++){
            mat[i + j*size] = (i*2+j)%(100);
        }
    }
    return 0;

}

int printMatrix(float* mat, int size){
#ifdef PRINT
    for(int i = 0; i < size; i++){
        for(int j = 0; j < size; j++){
            printf("%2.2f ", mat[i + j*size]);
        }
        printf("\n");
    }
    printf("\n");
#endif
    return 0;
}

int testTranspose(float* mat, float* mat_t, int size){
    for(int i = 0; i < size; i++){
        for(int j = 0; j < size; j++){
            if(mat[i + j*size] != mat_t[j + i*size]){
                printf("Error at mat[%d, %d]\n", i, j);
                return -1;
            }   
        }
    }    
    PRINTF("Matrix transposed without errors\n");    
    return 0;
}


int block_benchmark(uint N){
    //give access to the gpu
    int mem_size = N * N * sizeof(float);
    float* mat = (float*) malloc(mem_size);
    float* mat_t = (float*) malloc(mem_size);
    memset(mat_t, 0, mem_size);
    initMatrix(mat, N);
    float *d_mat, *d_mat_t;
    hipError_t err;
    //int threads, blocks = 0;
    PRINTF("Allocating memory\n");
    if((err = hipMalloc((void**)&d_mat, mem_size)) != hipSuccess){
        printf("Error allocating memory for d_a: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    if((err = hipMalloc((void**)&d_mat_t, mem_size)) != hipSuccess){
        printf("Error allocating memory for d_mat_t: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    //cudaMalloc((void**)&d_mat_t, mem_size);
    PRINTF("Memory allocated\n");
    //copy data to gpu
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    if((err = hipMemcpy(d_mat, mat, N * N * sizeof(int), hipMemcpyHostToDevice)) != hipSuccess){
        printf("Error copying data to d_mat: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    PRINTF("Data copied\n");
    //setup grid and block size
    dim3 DimGrid = {N/TILE_SIZE, N/TILE_SIZE, 1};
    dim3 DimBlock = {TILE_SIZE, BLOCK_ROWS, 1};
    
    //call kernel as many times as needed
    //first a dummy kernel
    block_transpose<<<DimGrid, DimBlock>>>(d_mat, d_mat_t);
    hipEvent_t startK, stopK;
    hipEventCreate(&startK);
    hipEventCreate(&stopK);
    hipEventRecord(startK);
    for(int i = 0; i < TRANSPOSITIONS; i++){
        block_transpose<<<DimGrid, DimBlock>>>(d_mat, d_mat_t);
    }
    hipEventRecord(stopK);
    hipEventSynchronize(stopK);
    PRINTF("Kernel returned\n");

    //copy data back
    if((err = hipMemcpy(mat_t, d_mat_t, mem_size, hipMemcpyDeviceToHost)) != hipSuccess){
        printf("Error copying data to mat_t: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    //sync
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    
    float millisecondsK = 0;
    hipEventElapsedTime(&millisecondsK, startK, stopK);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float ogbs = 2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS / milliseconds;
    float kgbs = 2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS / millisecondsK;
    PRINTF("Operation Time: %11.2f ms\n", milliseconds);
    PRINTF("Throughput in GB/s: %7.2f\n", ogbs);
    PRINTF("Kernel Time: %11.2f ms\n", millisecondsK);
    PRINTF("Throughput in GB/s: %7.2f\n", kgbs);
    printf("%f, %f, %f, %f\n", milliseconds, ogbs, millisecondsK, kgbs);
    hipEventDestroy(startK);
    hipEventDestroy(stopK);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    //results
    printMatrix(mat_t, N);

    //test if the matrix is transposed
    testTranspose(mat, mat_t, N);  

    //free gpu resources
    hipFree(d_mat);
    hipFree(d_mat_t);
    free(mat);
    free(mat_t);
    return 0;
}

int basic_benchmark(uint N){
    //give access to the gpu
    int mem_size = N * N * sizeof(float);
    float* mat = (float*) malloc(mem_size);
    float* mat_t = (float*) malloc(mem_size);
    memset(mat_t, 0, mem_size);
    initMatrix(mat, N);
    float *d_mat, *d_mat_t;
    hipError_t err;
    //int threads, blocks = 0;
    PRINTF("Allocating memory\n");
    if((err = hipMalloc((void**)&d_mat, mem_size)) != hipSuccess){
        printf("Error allocating memory for d_a: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    if((err = hipMalloc((void**)&d_mat_t, mem_size)) != hipSuccess){
        printf("Error allocating memory for d_mat_t: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    //cudaMalloc((void**)&d_mat_t, mem_size);
    PRINTF("Memory allocated\n");

    //copy data to gpu
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    if((err = hipMemcpy(d_mat, mat, mem_size, hipMemcpyHostToDevice)) != hipSuccess){
        printf("Error copying data to d_mat: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    PRINTF("Data copied\n");
    //setup grid and block size
    dim3 gridB(N / B_TILE, N / B_TILE);
    dim3 blockB(B_TILE, B_ROWS);
    
    //call kernel as many times as needed
    //first a dummy kernel
    basic_transpose<<<gridB, blockB>>>(d_mat, d_mat_t, N);
    hipEvent_t startK, stopK;
    hipEventCreate(&startK);
    hipEventCreate(&stopK);
    hipEventRecord(startK);
    for(int i = 0; i < TRANSPOSITIONS; i++){
        basic_transpose<<<gridB, blockB>>>(d_mat, d_mat_t, N);
    }
    hipEventRecord(stopK);
    hipEventSynchronize(stopK);
    PRINTF("Kernel returned\n");

    //copy data back
    if((err = hipMemcpy(mat_t, d_mat_t, mem_size, hipMemcpyDeviceToHost)) != hipSuccess){
        printf("Error copying data to mat_t: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    //sync
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    
    float millisecondsK = 0;
    hipEventElapsedTime(&millisecondsK, startK, stopK);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float ogbs = 2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS / milliseconds;
    float kgbs = 2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS / millisecondsK;
    PRINTF("Operation Time: %11.2f ms\n", milliseconds);
    PRINTF("Throughput in GB/s: %7.2f\n", ogbs);
    PRINTF("Kernel Time: %11.2f ms\n", millisecondsK);
    PRINTF("Throughput in GB/s: %7.2f\n", kgbs);
    printf("%f, %f, %f, %f, ", milliseconds, ogbs, millisecondsK, kgbs);

    hipEventDestroy(startK);
    hipEventDestroy(stopK);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //test if the matrix is transposed
    PRINTF("basic results:\n");
    printMatrix(mat_t, N);
    testTranspose(mat, mat_t, N);  

    //reset output matrix
    memset(mat_t, 0, mem_size);
    //cudaMemset(d_mat_t, 0, mem_size);
    hipFree(d_mat);
    hipFree(d_mat_t);
    free(mat);
    free(mat_t);
    return 0;
}

int conflict_benchmark(uint N){
    //give access to the gpu
    int mem_size = N * N * sizeof(float);
    float* mat = (float*) malloc(mem_size);
    float* mat_t = (float*) malloc(mem_size);
    memset(mat_t, 0, mem_size);
    initMatrix(mat, N);
    float *d_mat, *d_mat_t;
    hipError_t err;
    //int threads, blocks = 0;
    PRINTF("Allocating memory\n");
    if((err = hipMalloc((void**)&d_mat, mem_size)) != hipSuccess){
        printf("Error allocating memory for d_a: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    if((err = hipMalloc((void**)&d_mat_t, mem_size)) != hipSuccess){
        printf("Error allocating memory for d_mat_t: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    //cudaMalloc((void**)&d_mat_t, mem_size);
    PRINTF("Memory allocated\n");
    //copy data to gpu
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    if((err = hipMemcpy(d_mat, mat, N * N * sizeof(int), hipMemcpyHostToDevice)) != hipSuccess){
        printf("Error copying data to d_mat: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    PRINTF("Data copied\n");
    //setup grid and block size
    dim3 DimGrid = {N/TILE_SIZE, N/TILE_SIZE, 1};
    dim3 DimBlock = {TILE_SIZE, BLOCK_ROWS, 1};
    
    //call kernel as many times as needed
    //first a dummy kernel
    conflict_transpose<<<DimGrid, DimBlock>>>(d_mat, d_mat_t);
    hipEvent_t startK, stopK;
    hipEventCreate(&startK);
    hipEventCreate(&stopK);
    hipEventRecord(startK);
    for(int i = 0; i < TRANSPOSITIONS; i++){
        conflict_transpose<<<DimGrid, DimBlock>>>(d_mat, d_mat_t);
    }
    hipEventRecord(stopK);
    hipEventSynchronize(stopK);
    PRINTF("Kernel returned\n");

    //copy data back
    if((err = hipMemcpy(mat_t, d_mat_t, mem_size, hipMemcpyDeviceToHost)) != hipSuccess){
        printf("Error copying data to mat_t: %s\n", hipGetErrorString(err));
        printf("Line: %d\n", __LINE__);
        return -1;
    }
    //sync
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    
    float millisecondsK = 0;
    hipEventElapsedTime(&millisecondsK, startK, stopK);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    PRINTF("Operation Time: %11.2f ms\n", milliseconds);
    float ogbs = 2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS / milliseconds;
    float kgbs = 2 * N * N * sizeof(float) * 1e-6 * TRANSPOSITIONS / millisecondsK;
    PRINTF("Throughput in GB/s: %7.2f\n", ogbs);
    PRINTF("Kernel Time: %11.2f ms\n", millisecondsK);
    PRINTF("Throughput in GB/s: %7.2f\n", kgbs);
    printf("%f, %f, %f, %f, ", milliseconds, ogbs, millisecondsK, kgbs);

    hipEventDestroy(startK);
    hipEventDestroy(stopK);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    //results
    printMatrix(mat_t, N);

    //test if the matrix is transposed
    testTranspose(mat, mat_t, N);  

    //free gpu resources
    hipFree(d_mat);
    hipFree(d_mat_t);
    free(mat);
    free(mat_t);
    return 0;
}